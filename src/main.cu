#include "hip/hip_runtime.h"
#define STB_IMAGE_IMPLEMENTATION
#include <hiprand/hiprand_kernel.h>
#include <float.h>
#include <iostream>
#include <math.h>
#include <hip/hip_math_constants.h>
#include <time.h>

#include "bvh.cuh"
#include "camera.cuh"
#include "hittable_list.cuh"
#include "image_io.h"
#include "material.cuh"
#include "perlin.cuh"
#include "ray.cuh"
#include "quad.cuh"
#include "sphere.cuh"
#include "texture.cuh"
#include "vec3.cuh"

#define checkCudaErrors(val) check_cuda((val), #val, __FILE__, __LINE__)
void check_cuda(hipError_t result, char const *const func, const char *const file, int const line)
{
    if (result)
    {
        std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << "at " <<
        file << ":" << line << " '" << func << "' \n";

        // Make sure we call CUDA Device Reset before exiting
        hipDeviceReset();
        exit(99);
    }
}

__device__ inline float apply_gamma(float c, float gamma)
{
    if (gamma == 1.0f) return c;
    float inv = 1.0f / gamma;
    return powf(fmaxf(c, 0.0f), inv);
}

__device__ vec3 color(const ray& r0,
                      const vec3& background,
                      bool gradient_bg,
                      hittable **world,
                      hiprandState *local_rand_state)
{
    ray  cur_ray        = r0;
    vec3 throughput     = vec3(1,1,1);
    vec3 radiance       = vec3(0,0,0);

    for (int bounce = 0; bounce < 50; ++bounce) 
    {
        hit_record rec;
        if (!(*world)->hit(cur_ray, 0.001f, FLT_MAX, rec)) {
            // miss: add background
            vec3 bg = background;
            if (gradient_bg) 
            {
                vec3 unit_direction = unit_vector(cur_ray.direction());
                float t = 0.5f*(unit_direction.y() + 1.0f);
                bg = (1.0f - t)*vec3(1.0, 1.0, 1.0) + t*vec3(0.5, 0.7, 1.0);
            }
            radiance += throughput * bg;
            break;
        }

        // add emission at this hit
        radiance += throughput * rec.mat_ptr->emitted(rec.u, rec.v, rec.p);

        // scatter
        ray  scattered;
        vec3 attenuation;
        if (!rec.mat_ptr->scatter(cur_ray, rec, attenuation, scattered, local_rand_state)) 
        {
            // light or absorbing surface: we’re done
            break;
        }

        throughput *= attenuation;
        cur_ray = scattered;
    }

    return radiance;
}

__global__ void rand_init(hiprandState *rand_state) 
{
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        hiprand_init(1984, 0, 0, rand_state);
    }
}

__global__ void render_init(int max_x, int max_y, hiprandState *rand_state)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if((i >= max_x) || (j >= max_y)) return;
    int pixel_index = j*max_x + i;

    // Same seed for each thread
    hiprand_init(1984+pixel_index, 0, 0, &rand_state[pixel_index]);
}

__global__ void render(vec3 *fb, int max_x, int max_y, int ns, float gamma,
                       camera **cam, hittable **world, hiprandState *rand_state,
                       vec3 background, int use_gradient_bg)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if ((i >= max_x) || (j >= max_y)) return;

    int pixel_index = j*max_x + i;
    hiprandState local_rand_state = rand_state[pixel_index];

    vec3 col(0,0,0);
    for (int s = 0; s < ns; s++) 
    {
        float u = float(i + hiprand_uniform(&local_rand_state)) / float(max_x);
        float v = float(j + hiprand_uniform(&local_rand_state)) / float(max_y);
        ray r = (*cam)->get_ray(u, v, &local_rand_state);
        col += color(r, background, use_gradient_bg != 0, world, &local_rand_state);
    }
    rand_state[pixel_index] = local_rand_state;

    col /= float(ns);
    col[0] = apply_gamma(col[0], gamma);
    col[1] = apply_gamma(col[1], gamma);
    col[2] = apply_gamma(col[2], gamma);
    fb[pixel_index] = col;
}

// ----- Configurable scene parameters -----
// Random helper (same as you had)
#define RND (hiprand_uniform(&local_rand_state))

// Grid config from the book example
#define GRID_MIN   -11
#define GRID_MAX    11
#define GRID_SIZE   (GRID_MAX - GRID_MIN)     // 22
#define TOTAL_SMALL (GRID_SIZE * GRID_SIZE)   // 22*22 = 484

// 1 ground + 3 big spheres + all small spheres
#define NUM_OBJECTS (1 + 3 + TOTAL_SMALL)

__global__ void create_world_bouncing(hittable **d_list, hittable **d_world, camera **d_camera,
                             int nx, int ny, hiprandState *rand_state)
{
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        hiprandState local_rand_state = *rand_state;
        int i = 0;

        // Ground (checkered)
        texture* checker = new checker_texture(0.32f,
            new solid_color(vec3(0.2f, 0.3f, 0.1f)),
            new solid_color(vec3(0.9f, 0.9f, 0.9f)));

        d_list[i++] = new sphere(vec3(0.0f, -1000.0f, -1.0f), 1000.0f,
                                new lambertian(checker));


        // Random small spheres on a grid
        for (int a = GRID_MIN; a < GRID_MAX; a++) {
            for (int b = GRID_MIN; b < GRID_MAX; b++) {
                float choose_mat = RND;
                vec3 center(a + 0.9f * RND, 0.2f, b + 0.9f * RND); // 0.9 like the book

                if (choose_mat < 0.8f) {
                    // Diffuse — MOVING
                    vec3 albedo(RND*RND, RND*RND, RND*RND);

                    // random velocity in each axis
                    vec3 vel(0.0f, 0.5f*RND, 0.25f*(RND - 0.5f));
                    vec3 center2 = center + vel;
                    d_list[i++] = new sphere(center, center2, 0.2f, new lambertian(albedo));
                } else if (choose_mat < 0.95f) {
                    // Metal with fuzz (static)
                    vec3 albedo(0.5f*(1.0f+RND), 0.5f*(1.0f+RND), 0.5f*(1.0f+RND));
                    float fuzz = 0.5f * RND;
                    d_list[i++] = new sphere(center, 0.2f, new metal(albedo, fuzz));
                } else {
                    // Dielectric (static)
                    d_list[i++] = new sphere(center, 0.2f, new dielectric(1.5f));
                }
            }
        }

        // Three big spheres (static)
        d_list[i++] = new sphere(vec3( 0.0f, 1.0f,  0.0f), 1.0f, new dielectric(1.5f));
        d_list[i++] = new sphere(vec3(-4.0f, 1.0f,  0.0f), 1.0f, new lambertian(vec3(0.4f, 0.2f, 0.1f)));
        d_list[i++] = new sphere(vec3( 4.0f, 1.0f,  0.0f), 1.0f, new metal(vec3(0.7f, 0.6f, 0.5f), 0.0f));

        *rand_state = local_rand_state;
        *d_world = new bvh_node(d_list, 0, i);

        // Camera — add shutter times [0,1]
        vec3 lookfrom(13.0f, 2.0f, 3.0f);
        vec3 lookat (0.0f,  0.0f, 0.0f);
        vec3 vup(0.0f, 1.0f, 0.0f);
        float dist_to_focus = (lookfrom - lookat).length();
        float aperture = 0.1f;

        *d_camera = new camera(lookfrom, lookat, vup,
                               30.0f, float(nx)/float(ny),
                               aperture, dist_to_focus,
                               /*time0=*/0.0, /*time1=*/1.0);
    }
}

__global__ void create_world_checker(hittable **d_list, hittable **d_world, camera **d_camera,
                                     int nx, int ny, hiprandState *rand_state)
{
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        // (RNG not strictly needed here, but keep the pattern)
        hiprandState local_rand_state = *rand_state;
        int i = 0;

        // One shared checker texture + lambertian
        texture* checker = new checker_texture(0.32f,
            new solid_color(vec3(0.2f, 0.3f, 0.1f)),
            new solid_color(vec3(0.9f, 0.9f, 0.9f)));
        material* lam = new lambertian(checker);

        // Two big spheres (y = ±10), like the book’s “checkered_spheres”
        d_list[i++] = new sphere(vec3(0,-10,0), 10.0f, lam);
        d_list[i++] = new sphere(vec3(0, 10,0), 10.0f, lam);

        *d_world = new bvh_node(d_list, 0, i);

        // Camera (pinhole)
        vec3 lookfrom(13.0f, 2.0f, 3.0f);
        vec3 lookat (0.0f, 0.0f, 0.0f);
        vec3 vup(0.0f, 1.0f, 0.0f);
        float dist_to_focus = 10.0f;
        float aperture = 0.0f;

        *d_camera = new camera(lookfrom, lookat, vup,
                               20.0f, float(nx)/float(ny),
                               aperture, dist_to_focus,
                               0.0, 1.0);

        *rand_state = local_rand_state;
    }
}

__global__ void create_world_earth(hittable **d_list, hittable **d_world, camera **d_camera,
                                   int nx, int ny, DeviceImage earth_img)
{
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        int i = 0;

        // Textured earth sphere at the origin
        texture*  earth_tex  = new image_texture(earth_img);
        material* earth_lam  = new lambertian(earth_tex);
        d_list[i++] = new sphere(vec3(0,0,0), 2.0f, earth_lam);

        // Wrap in BVH (okay even for 1 object, matches your other scenes)
        *d_world = new bvh_node(d_list, 0, i);

        // Camera (pinhole)
        vec3 lookfrom(0.0f, 0.0f, 12.0f);
        vec3 lookat  (0.0f, 0.0f,  0.0f);
        vec3 vup     (0.0f, 1.0f,  0.0f);
        float dist_to_focus = 12.0f;
        float aperture      = 0.0f;

        *d_camera = new camera(lookfrom, lookat, vup,
                               20.0f, float(nx)/float(ny),
                               aperture, dist_to_focus,
                               0.0, 1.0);
    }
}

__global__ void create_world_perlin(hittable **d_list, hittable **d_world, camera **d_camera,
                                    int nx, int ny, float scale)
{
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        int i = 0;

        texture* pertext = new noise_texture(scale);
        material* lam    = new lambertian(pertext);

        d_list[i++] = new sphere(vec3(0,-1000,0), 1000.f, lam);
        d_list[i++] = new sphere(vec3(0,     2,0),    2.f, lam);

        *d_world = new bvh_node(d_list, 0, i);

        vec3 lookfrom(13,2,3), lookat(0,0,0), vup(0,1,0);
        *d_camera = new camera(lookfrom, lookat, vup,
                               20.0f, float(nx)/float(ny),
                               0.0f, 10.0f, 0.0, 1.0);
    }
}

__global__ void create_world_quads(hittable **d_list, hittable **d_world, camera **d_camera,
                                   int nx, int ny)
{
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        int i = 0;

        // Materials
        material* left_red     = new lambertian(vec3(1.0f, 0.2f, 0.2f));
        material* back_green   = new lambertian(vec3(0.2f, 1.0f, 0.2f));
        material* right_blue   = new lambertian(vec3(0.2f, 0.2f, 1.0f));
        material* upper_orange = new lambertian(vec3(1.0f, 0.5f, 0.0f));
        material* lower_teal   = new lambertian(vec3(0.2f, 0.8f, 0.8f));

        // Quads (same geometry as your serial version)
        d_list[i++] = new quad(vec3(-3,-2, 5), vec3(0, 0,-4), vec3(0, 4, 0), left_red);
        d_list[i++] = new quad(vec3(-2,-2, 0), vec3(4, 0, 0), vec3(0, 4, 0), back_green);
        d_list[i++] = new quad(vec3( 3,-2, 1), vec3(0, 0, 4), vec3(0, 4, 0), right_blue);
        d_list[i++] = new quad(vec3(-2, 3, 1), vec3(4, 0, 0), vec3(0, 0, 4), upper_orange);
        d_list[i++] = new quad(vec3(-2,-3, 5), vec3(4, 0, 0), vec3(0, 0,-4), lower_teal);

        *d_world = new bvh_node(d_list, 0, i);

        vec3 lookfrom(0,0,9), lookat(0,0,0), vup(0,1,0);
        *d_camera = new camera(lookfrom, lookat, vup,
                               80.0f, float(nx)/float(ny),
                               0.0f, 10.0f, 0.0, 1.0);
    }
}

__global__ void create_world_simple_light(hittable **d_list, hittable **d_world, camera **d_camera,
                                          int nx, int ny)
{
    if (threadIdx.x || blockIdx.x) return;
    int i = 0;

    // --- Perlin (book uses scale=4) ---
    texture* pertex = new noise_texture(4.0f);          // requires noise_texture in texture.cuh/perlin.cuh
    material* perlam = new lambertian(pertex);          // lambertian(texture*) ctor takes ownership

    // Ground + floating sphere use Perlin
    d_list[i++] = new sphere(vec3(0,-1000,0), 1000.f, perlam);
    d_list[i++] = new sphere(vec3(0,2,0),        2.f, new lambertian(new noise_texture(4.0f)));

    // Lights (don’t share one instance unless one of the leaves passes owns=false)
    material* light1 = new diffuse_light(vec3(4,4,4));
    material* light2 = new diffuse_light(vec3(4,4,4));
    d_list[i++] = new sphere(vec3(0,7,0), 2.f,  light1);
    d_list[i++] = new quad  (vec3(3,1,-2), vec3(2,0,0), vec3(0,2,0), light2);

    *d_world = new bvh_node(d_list, 0, i);

    // Camera (same as your current one)
    vec3 lookfrom(26,3,6), lookat(0,2,0), vup(0,1,0);
    float dist_to_focus = (lookfrom - lookat).length();
    *d_camera = new camera(lookfrom, lookat, vup,
                           20.0f, float(nx)/float(ny),
                           0.0f, dist_to_focus,
                           0.0, 1.0);
}

__global__ void create_world_cornell(hittable **d_list, hittable **d_world, camera **d_camera,
                                     int nx, int ny)
{
    if (threadIdx.x || blockIdx.x) return;
    int i = 0;

    // Only 3 lambertian materials
    material* red    = new lambertian(vec3(.65f,.05f,.05f));
    material* green  = new lambertian(vec3(.12f,.45f,.15f));
    material* white  = new lambertian(vec3(.73f,.73f,.73f));   // reuse everywhere
    material* light  = new diffuse_light(vec3(15.f,15.f,15.f));

    // Cornell walls (inward-facing quads)
    d_list[i++] = new quad(vec3(0,0,0),       vec3(0,555,0),  vec3(0,0,555),  green,  true); // left
    d_list[i++] = new quad(vec3(555,0,555),   vec3(0,555,0),  vec3(0,0,-555), red,    true); // right
    d_list[i++] = new quad(vec3(0,0,0),       vec3(555,0,0),  vec3(0,0,555),  white,  true); // floor
    d_list[i++] = new quad(vec3(0,555,555),   vec3(555,0,0),  vec3(0,0,-555), white,  true); // ceiling
    d_list[i++] = new quad(vec3(555,0,555),   vec3(-555,0,0), vec3(0,555,0),  white,  true); // back
    d_list[i++] = new quad(vec3(213,554,227), vec3(130,0,0),  vec3(0,0,105),  light,  true); // light

    // ---- Instanced boxes ----
    // Build two *properly sized* prototypes (same geometry type, different height).
    hittable* proto_short = make_box(vec3(0,0,0), vec3(165,165,165), white);
    hittable* proto_tall  = make_box(vec3(0,0,0), vec3(165,330,165), white);

    // Place them using the canonical Cornell transforms.
    d_list[i++] = new translate(new rotate_y(proto_short, -18.f), vec3(130.f, 0.f,  65.f));
    d_list[i++] = new translate(new rotate_y(proto_tall,   15.f), vec3(265.f, 0.f, 295.f));

    *d_world = new bvh_node(d_list, 0, i);

    // Camera
    vec3 lookfrom(278,278,-800), lookat(278,278,0), vup(0,1,0);
    float dist_to_focus = (lookfrom - lookat).length();
    *d_camera = new camera(lookfrom, lookat, vup,
                           40.0f, float(nx)/float(ny),
                           0.0f, dist_to_focus,
                           0.0, 1.0);
}

__global__ void free_world(hittable **d_list, int count,
                           hittable **d_world,
                           camera   **d_camera)
{
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        // 1) Delete all leaves (their virtual dtors free owned materials)
        for (int i = 0; i < count; ++i)
            delete d_list[i];

        // 2) Delete BVH internal nodes (dtor skips leaves by design)
        delete *d_world;

        // 3) Delete camera
        delete *d_camera;
    }
}

void bouncing_spheres()
{
    int nx = 1200;
    int ny = 600;
    int ns = 500;
    float gamma = 2.2f;
    int tx = 8;
    int ty = 8;

    // Increase per thread call stack size and device heap
    // Temporary workaround since bvh is still recursive 
    hipDeviceSetLimit(hipLimitStackSize,      16384);        // 16 KB
    hipDeviceSetLimit(hipLimitMallocHeapSize, 64*1024*1024); // 64 MB device heap

    std::cerr << "Rendering a " << nx << "x" << ny << " image ";
    std::cerr << "in " << tx << "x" << ty << " blocks.\n";

    int num_pixels = nx * ny;
    size_t fb_size = num_pixels * sizeof(vec3);

    // Allocate frame buffer
    vec3 *fb;
    checkCudaErrors(hipMallocManaged((void **)&fb, fb_size));

    // Allocate random state
    hiprandState *d_rand_state;
    checkCudaErrors(hipMalloc((void **)&d_rand_state, num_pixels*sizeof(hiprandState)));
    hiprandState *d_rand_state2;
    checkCudaErrors(hipMalloc((void **)&d_rand_state2, 1*sizeof(hiprandState)));

    // we need that 2nd random state to be initialized for the world creation
    rand_init<<<1,1>>>(d_rand_state2);

    // Make camera and world with hittables
    camera **d_camera;
    checkCudaErrors(hipMalloc((void **)&d_camera, sizeof(camera *)));
    hittable **d_list;
    int num_hitables = 22*22 + 1 + 3;
    checkCudaErrors(hipMalloc((void **)&d_list, num_hitables * sizeof(hittable *)));
    hittable **d_world;
    checkCudaErrors(hipMalloc((void **)&d_world, sizeof(hittable *)));
    create_world_bouncing<<<1,1>>>(d_list, d_world, d_camera, nx, ny, d_rand_state2);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    clock_t start, stop;
    start = clock();
    // Render buffer
    dim3 blocks(nx/tx+1, ny/ty+1);
    dim3 threads(tx, ty);
    render_init<<<blocks, threads>>>(nx, ny, d_rand_state);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    render<<<blocks, threads>>>(fb, nx, ny,  ns, gamma, d_camera, d_world, d_rand_state, vec3(0,0,0), 1);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    stop = clock();
    double timer_seconds = ((double(stop - start)) / CLOCKS_PER_SEC);
    std::cerr << "took " << timer_seconds << " seconds.\n";

    // Output frame buffer as image
    std::cout << "P3\n" << nx << " " << ny << "\n255\n";

    for (int j = ny-1; j >= 0; j--)
    {
        for (int i = 0; i < nx; i++)
        {
            size_t pixel_index = j*nx + i;
            int ir = int(255.99*fb[pixel_index].r());
            int ig = int(255.99*fb[pixel_index].g());
            int ib = int(255.99*fb[pixel_index].b());
            std::cout << ir << " " << ig << " " << ib << "\n";
        }
    }
    
    // Clean up
    checkCudaErrors(hipDeviceSynchronize());                 // make sure render finished

    free_world<<<1,1>>>(d_list, num_hitables, d_world, d_camera);   // NOTE: count is 2nd arg
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());                 // wait for device-side deletes

    checkCudaErrors(hipFree(d_camera));
    checkCudaErrors(hipFree(d_world));
    checkCudaErrors(hipFree(d_list));
    checkCudaErrors(hipFree(d_rand_state));
    checkCudaErrors(hipFree(fb));

    // Useful for cuda-memcheck --leak-check full
    hipDeviceReset();
}

int checkered_spheres() {
    int nx = 1200, ny = 600, ns = 500;
    float gamma = 2.2f;
    int tx = 8, ty = 8;

    hipDeviceSetLimit(hipLimitStackSize,      16384);
    hipDeviceSetLimit(hipLimitMallocHeapSize, 64*1024*1024);

    int num_pixels = nx * ny;
    size_t fb_size = num_pixels * sizeof(vec3);

    vec3 *fb;                      checkCudaErrors(hipMallocManaged((void **)&fb, fb_size));
    hiprandState *d_rand_state;     checkCudaErrors(hipMalloc((void **)&d_rand_state,  num_pixels*sizeof(hiprandState)));
    hiprandState *d_rand_state2;    checkCudaErrors(hipMalloc((void **)&d_rand_state2, 1*sizeof(hiprandState)));
    rand_init<<<1,1>>>(d_rand_state2);

    camera **d_camera;             checkCudaErrors(hipMalloc((void **)&d_camera, sizeof(camera *)));
    int num_hitables = 2;          // two big spheres
    hittable **d_list;             checkCudaErrors(hipMalloc((void **)&d_list, num_hitables * sizeof(hittable *)));
    hittable **d_world;            checkCudaErrors(hipMalloc((void **)&d_world, sizeof(hittable *)));

    create_world_checker<<<1,1>>>(d_list, d_world, d_camera, nx, ny, d_rand_state2);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    dim3 blocks(nx/tx+1, ny/ty+1), threads(tx, ty);
    render_init<<<blocks, threads>>>(nx, ny, d_rand_state);
    checkCudaErrors(hipDeviceSynchronize());
    render<<<blocks, threads>>>(fb, nx, ny, ns, gamma, d_camera, d_world, d_rand_state, vec3(0,0,0), 1);
    checkCudaErrors(hipDeviceSynchronize());

    std::cout << "P3\n" << nx << " " << ny << "\n255\n";
    for (int j = ny-1; j >= 0; --j) for (int i = 0; i < nx; ++i) {
        size_t k = j*nx + i;
        int ir = int(255.99f*fb[k].r());
        int ig = int(255.99f*fb[k].g());
        int ib = int(255.99f*fb[k].b());
        std::cout << ir << " " << ig << " " << ib << "\n";
    }

    // --- Clean up ---
    checkCudaErrors(hipDeviceSynchronize());                 // make sure render finished

    free_world<<<1,1>>>(d_list, num_hitables, d_world, d_camera);   // NOTE: count is 2nd arg
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());                 // wait for device-side deletes

    checkCudaErrors(hipFree(d_camera));
    checkCudaErrors(hipFree(d_world));
    checkCudaErrors(hipFree(d_list));
    checkCudaErrors(hipFree(d_rand_state));
    checkCudaErrors(hipFree(fb));
    hipDeviceReset();
    return 0;
}

int earth() {
    // Render params (match your other scenes)
    int nx = 1200;
    int ny = 600;
    int ns = 500;
    float gamma = 2.2f;
    int tx = 8, ty = 8;

    // Device limits (you already do this elsewhere too)
    hipDeviceSetLimit(hipLimitStackSize,      16384);
    hipDeviceSetLimit(hipLimitMallocHeapSize, 64*1024*1024);

    // Load earth texture on HOST, upload to device
    unsigned char* d_pixels = nullptr;
    DeviceImage earth_img   = load_image_to_device("textures/earthmap.jpg", &d_pixels);
    if (!earth_img.valid()) {
        std::cerr << "Failed to load earthmap.jpg\n";
        return 1;
    }

    // Framebuffer
    int num_pixels = nx * ny;
    size_t fb_size = num_pixels * sizeof(vec3);
    vec3 *fb; checkCudaErrors(hipMallocManaged((void **)&fb, fb_size));

    // RNG
    hiprandState *d_rand_state;  checkCudaErrors(hipMalloc((void **)&d_rand_state,  num_pixels*sizeof(hiprandState)));
    hiprandState *d_rand_state2; checkCudaErrors(hipMalloc((void **)&d_rand_state2, 1*sizeof(hiprandState)));
    rand_init<<<1,1>>>(d_rand_state2);

    // Scene allocations
    camera **d_camera; checkCudaErrors(hipMalloc((void **)&d_camera, sizeof(camera *)));
    int num_hitables = 1;       // just the globe
    hittable **d_list;  checkCudaErrors(hipMalloc((void **)&d_list,  num_hitables * sizeof(hittable *)));
    hittable **d_world; checkCudaErrors(hipMalloc((void **)&d_world, sizeof(hittable *)));

    // Build world on device
    create_world_earth<<<1,1>>>(d_list, d_world, d_camera, nx, ny, earth_img);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    // Render
    dim3 blocks(nx/tx+1, ny/ty+1), threads(tx, ty);
    render_init<<<blocks, threads>>>(nx, ny, d_rand_state);
    checkCudaErrors(hipDeviceSynchronize());
    render<<<blocks, threads>>>(fb, nx, ny, ns, gamma, d_camera, d_world, d_rand_state, vec3(0,0,0), 1);
    checkCudaErrors(hipDeviceSynchronize());

    // Output PPM
    std::cout << "P3\n" << nx << " " << ny << "\n255\n";
    for (int j = ny-1; j >= 0; --j) {
        for (int i = 0; i < nx; ++i) {
            size_t k = j*nx + i;
            int ir = int(255.99f*fb[k].r());
            int ig = int(255.99f*fb[k].g());
            int ib = int(255.99f*fb[k].b());
            std::cout << ir << " " << ig << " " << ib << "\n";
        }
    }

    // Cleanup
    checkCudaErrors(hipDeviceSynchronize());                 // make sure render finished

    free_world<<<1,1>>>(d_list, num_hitables, d_world, d_camera);   // NOTE: count is 2nd arg
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());                 // wait for device-side deletes

    checkCudaErrors(hipFree(d_camera));
    checkCudaErrors(hipFree(d_world));
    checkCudaErrors(hipFree(d_list));
    checkCudaErrors(hipFree(d_rand_state));
    checkCudaErrors(hipFree(fb));

    // Free device pixels for the earth texture
    free_device_image(earth_img);

    hipDeviceReset();
    return 0;
}

int perlin() {
    int nx = 1200, ny = 600, ns = 500;
    float gamma = 2.2f;
    int tx = 8, ty = 8;

    hipDeviceSetLimit(hipLimitStackSize,      16384);
    hipDeviceSetLimit(hipLimitMallocHeapSize, 64*1024*1024);

    int num_pixels = nx * ny;
    size_t fb_size = num_pixels * sizeof(vec3);

    vec3 *fb;                      checkCudaErrors(hipMallocManaged((void **)&fb, fb_size));
    hiprandState *d_rand_state;     checkCudaErrors(hipMalloc((void **)&d_rand_state,  num_pixels*sizeof(hiprandState)));
    hiprandState *d_rand_state2;    checkCudaErrors(hipMalloc((void **)&d_rand_state2, 1*sizeof(hiprandState)));
    rand_init<<<1,1>>>(d_rand_state2);

    camera **d_camera;             checkCudaErrors(hipMalloc((void **)&d_camera, sizeof(camera *)));
    int num_hitables = 2;          // ground + sphere
    hittable **d_list;             checkCudaErrors(hipMalloc((void **)&d_list,  num_hitables * sizeof(hittable *)));
    hittable **d_world;            checkCudaErrors(hipMalloc((void **)&d_world, sizeof(hittable *)));

    const float scale = 4.0f;      // tweak to taste (like the book)
    create_world_perlin<<<1,1>>>(d_list, d_world, d_camera, nx, ny, scale);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    dim3 blocks(nx/tx+1, ny/ty+1), threads(tx, ty);
    render_init<<<blocks, threads>>>(nx, ny, d_rand_state);
    checkCudaErrors(hipDeviceSynchronize());
    render<<<blocks, threads>>>(fb, nx, ny, ns, gamma, d_camera, d_world, d_rand_state, vec3(0,0,0), 1);
    checkCudaErrors(hipDeviceSynchronize());

    std::cout << "P3\n" << nx << " " << ny << "\n255\n";
    for (int j = ny-1; j >= 0; --j) for (int i = 0; i < nx; ++i) {
        size_t k = j*nx + i;
        int ir = int(255.99f*fb[k].r());
        int ig = int(255.99f*fb[k].g());
        int ib = int(255.99f*fb[k].b());
        std::cout << ir << " " << ig << " " << ib << "\n";
    }

    // --- Clean up ---
    checkCudaErrors(hipDeviceSynchronize());                 // make sure render finished

    free_world<<<1,1>>>(d_list, num_hitables, d_world, d_camera);   // NOTE: count is 2nd arg
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());                 // wait for device-side deletes

    checkCudaErrors(hipFree(d_camera));
    checkCudaErrors(hipFree(d_world));
    checkCudaErrors(hipFree(d_list));
    checkCudaErrors(hipFree(d_rand_state));
    checkCudaErrors(hipFree(fb));
    hipDeviceReset();
    return 0;
}

int quads_scene() {
    int nx = 1200, ny = 600, ns = 500;
    float gamma = 2.2f;
    int tx = 8, ty = 8;

    hipDeviceSetLimit(hipLimitStackSize,      16384);
    hipDeviceSetLimit(hipLimitMallocHeapSize, 64*1024*1024);

    int num_pixels = nx * ny;
    size_t fb_size = num_pixels * sizeof(vec3);

    vec3 *fb;                   checkCudaErrors(hipMallocManaged((void**)&fb, fb_size));
    hiprandState *d_rand_state;  checkCudaErrors(hipMalloc((void**)&d_rand_state,  num_pixels*sizeof(hiprandState)));
    hiprandState *d_rand_state2; checkCudaErrors(hipMalloc((void**)&d_rand_state2, 1*sizeof(hiprandState)));
    rand_init<<<1,1>>>(d_rand_state2);

    camera **d_camera; checkCudaErrors(hipMalloc((void**)&d_camera, sizeof(camera*)));
    int num_hitables = 5;
    hittable **d_list; checkCudaErrors(hipMalloc((void**)&d_list,  num_hitables*sizeof(hittable*)));
    hittable **d_world;checkCudaErrors(hipMalloc((void**)&d_world, sizeof(hittable*)));

    create_world_quads<<<1,1>>>(d_list, d_world, d_camera, nx, ny);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    dim3 blocks(nx/tx+1, ny/ty+1), threads(tx, ty);
    render_init<<<blocks, threads>>>(nx, ny, d_rand_state);
    checkCudaErrors(hipDeviceSynchronize());
    render<<<blocks, threads>>>(fb, nx, ny, ns, gamma, d_camera, d_world, d_rand_state, vec3(0,0,0), 1);
    checkCudaErrors(hipDeviceSynchronize());

    std::cout << "P3\n" << nx << " " << ny << "\n255\n";
    for (int j = ny-1; j >= 0; --j) for (int i = 0; i < nx; ++i) {
        size_t k = j*nx + i;
        int ir = int(255.99f*fb[k].r());
        int ig = int(255.99f*fb[k].g());
        int ib = int(255.99f*fb[k].b());
        std::cout << ir << " " << ig << " " << ib << "\n";
    }

    // --- Clean up ---
    checkCudaErrors(hipDeviceSynchronize());                 // make sure render finished

    free_world<<<1,1>>>(d_list, num_hitables, d_world, d_camera);   // NOTE: count is 2nd arg
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());                 // wait for device-side deletes

    checkCudaErrors(hipFree(d_camera));
    checkCudaErrors(hipFree(d_world));
    checkCudaErrors(hipFree(d_list));
    checkCudaErrors(hipFree(d_rand_state));
    checkCudaErrors(hipFree(fb));
    hipDeviceReset();
    return 0;
}

int simple_light() 
{
    // image / render params
    int nx = 1200;
    int ny = 600;
    int ns = 500;
    float gamma = 2.2f;
    int tx = 8, ty = 8;

    // device limits (same as your other scenes)
    hipDeviceSetLimit(hipLimitStackSize,      16384);
    hipDeviceSetLimit(hipLimitMallocHeapSize, 64*1024*1024);

    // frame buffer
    int num_pixels = nx * ny;
    size_t fb_size = num_pixels * sizeof(vec3);
    vec3 *fb; checkCudaErrors(hipMallocManaged((void**)&fb, fb_size));

    // RNG
    hiprandState *d_rand_state;  checkCudaErrors(hipMalloc((void**)&d_rand_state,  num_pixels*sizeof(hiprandState)));
    hiprandState *d_rand_state2; checkCudaErrors(hipMalloc((void**)&d_rand_state2, sizeof(hiprandState)));
    rand_init<<<1,1>>>(d_rand_state2);

    // scene storage
    camera   **d_camera; checkCudaErrors(hipMalloc((void**)&d_camera, sizeof(camera*)));
    const int num_hitables = 4; // ground + gray sphere + light sphere + light quad
    hittable **d_list;   checkCudaErrors(hipMalloc((void**)&d_list,   num_hitables * sizeof(hittable*)));
    hittable **d_world;  checkCudaErrors(hipMalloc((void**)&d_world,  sizeof(hittable*)));

    // build the scene on device
    create_world_simple_light<<<1,1>>>(d_list, d_world, d_camera, nx, ny);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    // render
    dim3 blocks(nx/tx+1, ny/ty+1), threads(tx,ty);
    render_init<<<blocks, threads>>>(nx, ny, d_rand_state);              checkCudaErrors(hipDeviceSynchronize());
    render     <<<blocks, threads>>>(fb, nx, ny, ns, gamma, d_camera, d_world, d_rand_state,
                                     /*background*/ vec3(0,0,0), /*use_gradient_bg*/ 0);
    checkCudaErrors(hipDeviceSynchronize());

    // output PPM
    std::cout << "P3\n" << nx << " " << ny << "\n255\n";
    for (int j = ny-1; j >= 0; --j) {
        for (int i = 0; i < nx; ++i) {
            const size_t k = j*nx + i;
            int ir = int(255.99f*fb[k].r());
            int ig = int(255.99f*fb[k].g());
            int ib = int(255.99f*fb[k].b());
            std::cout << ir << ' ' << ig << ' ' << ib << '\n';
        }
    }

    // --- Clean up ---
    checkCudaErrors(hipDeviceSynchronize());                 // make sure render finished

    free_world<<<1,1>>>(d_list, num_hitables, d_world, d_camera);   // NOTE: count is 2nd arg
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());                 // wait for device-side deletes

    checkCudaErrors(hipFree(d_camera));
    checkCudaErrors(hipFree(d_world));
    checkCudaErrors(hipFree(d_list));
    checkCudaErrors(hipFree(d_rand_state));
    checkCudaErrors(hipFree(fb));
    hipDeviceReset();
    return 0;
}

int cornell_box() 
{
    int nx = 600, ny = 600, ns = 1000;
    float gamma = 2.2f;
    int tx = 8, ty = 8;

    hipDeviceSetLimit(hipLimitStackSize,      16384);
    hipDeviceSetLimit(hipLimitMallocHeapSize, 64*1024*1024);

    int num_pixels = nx * ny;
    size_t fb_size = num_pixels * sizeof(vec3);
    vec3 *fb; checkCudaErrors(hipMallocManaged((void**)&fb, fb_size));

    hiprandState *d_rand_state;  checkCudaErrors(hipMalloc((void**)&d_rand_state,  num_pixels*sizeof(hiprandState)));
    hiprandState *d_rand_state2; checkCudaErrors(hipMalloc((void**)&d_rand_state2, sizeof(hiprandState)));
    rand_init<<<1,1>>>(d_rand_state2);

    camera **d_camera; checkCudaErrors(hipMalloc((void**)&d_camera, sizeof(camera*)));
    const int num_hitables = 6; // the 6 Cornell quads shown
    hittable **d_list;  checkCudaErrors(hipMalloc((void**)&d_list,  num_hitables * sizeof(hittable*)));
    hittable **d_world; checkCudaErrors(hipMalloc((void**)&d_world, sizeof(hittable*)));

    create_world_cornell<<<1,1>>>(d_list, d_world, d_camera, nx, ny);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    dim3 blocks(nx/tx+1, ny/ty+1), threads(tx,ty);
    render_init<<<blocks, threads>>>(nx, ny, d_rand_state);              checkCudaErrors(hipDeviceSynchronize());
    render     <<<blocks, threads>>>(fb, nx, ny, ns, gamma, d_camera, d_world, d_rand_state,
                                     /*background*/ vec3(0,0,0), /*use_gradient_bg*/ 0);
    checkCudaErrors(hipDeviceSynchronize());

    std::cout << "P3\n" << nx << " " << ny << "\n255\n";
    for (int j = ny-1; j >= 0; --j) for (int i = 0; i < nx; ++i) {
        size_t k = j*nx + i;
        int ir = int(255.99f*fb[k].r());
        int ig = int(255.99f*fb[k].g());
        int ib = int(255.99f*fb[k].b());
        std::cout << ir << ' ' << ig << ' ' << ib << '\n';
    }

    // --- Clean up ---
    checkCudaErrors(hipDeviceSynchronize());                 // make sure render finished

    free_world<<<1,1>>>(d_list, num_hitables, d_world, d_camera);   // NOTE: count is 2nd arg
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());                 // wait for device-side deletes

    checkCudaErrors(hipFree(d_camera));
    checkCudaErrors(hipFree(d_world));
    checkCudaErrors(hipFree(d_list));
    checkCudaErrors(hipFree(d_rand_state));
    checkCudaErrors(hipFree(fb));
    hipDeviceReset();
    return 0;
}

int main() 
{
    switch (7) 
    {
        case 1: bouncing_spheres();
        case 2: checkered_spheres();
        case 3: earth();
        case 4: perlin();
        case 5: quads_scene();
        case 6: simple_light();
        case 7: cornell_box();
    }
}
