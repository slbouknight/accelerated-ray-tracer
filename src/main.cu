
#include <hip/hip_runtime.h>
#include <iostream>
#include <time.h>

#define checkCudaErrors(val) check_cuda((val), #val, __FILE__, __LINE__)
void check_cuda(hipError_t result, char const *const func, const char *const file, int const line)
{
    if (result)
    {
        std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << "at " <<
        file << ":" << " '" << func << "' \n";

        // Make sure we call CUDA Device Reset before exiting
        hipDeviceReset();
        exit(99);
    }
}

__global__ void render(float *fb, int max_x, int max_y)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if((i >= max_x) || (j >= max_y)) return;
    int pixel_index = j*max_x*3 + i*3;
    fb[pixel_index + 0] = float(i) / max_x;
    fb[pixel_index + 1] = float(j) / max_y;
    fb[pixel_index + 2] = 0.2;

}

int main()
{
    int nx = 1200;
    int ny = 600;
    int tx = 8;
    int ty = 8;

    int num_pixels = nx * ny;
    size_t fb_size = 3 * num_pixels * sizeof(float);

    // Allocate frame buffer
    float *fb;
    checkCudaErrors(hipMallocManaged((void **)&fb, fb_size));

    // Render buffer
    dim3 blocks(nx/tx+1, ny/ty+1);
    dim3 threads(tx, ty);
    render<<<blocks, threads>>>(fb, nx, ny);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    // Output frame buffer as image
    std::cout << "P3\n" << nx << " " << ny << "\n255\n";

    for (int j = ny-1; j >= 0; j--)
    {
        for (int i = 0; i < nx; i++)
        {
            size_t pixel_index = j *3*nx + i*3;
            float r = fb[pixel_index + 0];
            float g = fb[pixel_index + 1];
            float b = fb[pixel_index + 2];
            int ir = int(255.999*r);
            int ig = int(255.999*g);
            int ib = int(255.999*b);
            std::cout << ir << " " << ig << " " << ib << "\n";
        }
    }
    checkCudaErrors(hipFree(fb));
}